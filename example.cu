#include "hip/hip_runtime.h"
/*******************************************************************************
 * CUDA Kernel Timer
 * Copyright (C) 2024 Yiltan Hassan Temucin
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *******************************************************************************/
#include "cuda_kernel_timer.cuh"
#include "stdio.h" // For printf

__global__ void example_kernel(long long int  requested_time_ns,
                               long long int *measured_time_ns)
{
  long long int start;
  long long int stop;

  start = get_globaltimer();

  do {
    stop = get_globaltimer();
  } while ((stop - start) < requested_time_ns);
  *measured_time_ns = (stop - start);
}


static float to_ms(long long int time_ns)
{
    return ((float) time_ns / (1000.0 * 1000.0));
}

int main()
{
  hipEvent_t start, stop;
  long long int requested_time_ns;
  long long int *measured_time_ns;
  float milliseconds;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMallocManaged(&measured_time_ns, sizeof(long long int));

  requested_time_ns = 100 * 1000 * 1000; // 1ms

  hipEventRecord(start);
  example_kernel<<<1,1>>>(requested_time_ns, measured_time_ns);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Requested %fms, Kernel Timer Measured %fms, CUDA Events Measured %fms\n",
         to_ms(requested_time_ns), to_ms(*measured_time_ns), milliseconds);
  fflush(stdout);

  return 0;
}
