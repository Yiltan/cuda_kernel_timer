#include "hip/hip_runtime.h"
/*******************************************************************************
 * CUDA Kernel Timer
 * Copyright (C) 2024 Yiltan Hassan Temucin
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *******************************************************************************/
#include "cuda_kernel_timer.cuh"
#include "stdio.h" // For printf

__global__ void example_kernel(long long int  requested_time_ns,
                               long long int *measured_time_ns)
{
  long long int start;
  long long int stop;

  start = read_globaltimer();

  do {
    stop = read_globaltimer();
  } while ((stop - start) < requested_time_ns);
  *measured_time_ns = (stop - start);
}


static float to_ms(long long int time_ns)
{
    return ((float) time_ns / (1000.0 * 1000.0));
}

int main()
{
  hipEvent_t start, stop;
  long long int requested_time_ns;
  long long int *measured_time_ns;
  float milliseconds;

  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipMallocManaged(&measured_time_ns, sizeof(long long int));

  requested_time_ns = 100 * 1000 * 1000; // 1ms

  hipEventRecord(start);
  example_kernel<<<1,1>>>(requested_time_ns, measured_time_ns);
  hipEventRecord(stop);

  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);

  printf("Requested %.6fms, Kernel Timer Measured %.6fms, CUDA Events Measured %.3fms\n",
         to_ms(requested_time_ns), /* 6DP for 1ns resolution (did not account for float rounding) */
         to_ms(*measured_time_ns), /* 6DP for 1ns resolution (did not account for float rounding) */
         milliseconds              /* 3DP for 0.5us resolution */
        );
  fflush(stdout);

  return 0;
}
